#include "hip/hip_runtime.h"

#include "Window.cuh"
#include "Render.cuh"
#include "Hittable.cuh"
#include "IniParser.h"

#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

int main()
{
    std::string sceneINIPath;
    std::cout << "Enter scene ini file path (default: scene.ini):" << std::endl;
    std::getline(std::cin, sceneINIPath);
    // default
    if (sceneINIPath.empty()) 
        sceneINIPath = "C:/Users/59409/source/repos/CudaRayTracing/CudaRayTracing/scene_ini/Scene3.ini";

    IniParser parser;
    parser.Parse(sceneINIPath);

    int nx = parser.camera.width;
    int ny = static_cast<int>(nx / 16.0 * 9.0);
    
    const int threadsNum = 16;
    dim3 threads(threadsNum, threadsNum);
    dim3 blocks((nx + threadsNum - 1) / threadsNum, (ny + threadsNum - 1) / threadsNum);
    
    Scene scene;
    scene.init(parser);

    Camera camera(nx, 16.0 / 9.0, parser.camera.background, parser.camera.lookFrom, parser.camera.lookAt, parser.camera.vFov);
    checkCudaErrors(hipMemcpy(scene.d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));


    Window app(nx, ny, &camera, &scene);
    
    uchar4* d_gBuffer;
    hipMalloc((void**)&d_gBuffer, nx * ny * sizeof(uchar4));

    if (app.Init())
    {
        bool preStats = false;
        double preTime = 0;
        double t = 0;
        clear <<< blocks, threads >>> (app.devicePtr, nx, ny);
        hipDeviceSynchronize();

        while (!app.Close())
        {
            if (app.PollInput())
                checkCudaErrors(hipMemcpy(scene.d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
            if (!app.paused || app.paused != preStats)
            {
                t = (double)glfwGetTime();
                int sampleCount = app.sampleCount;
                if (app.paused)
                {
                    std::cout << "Rendering " << app.sampleCount << " sample count..." << std::endl;
                    t = preTime;
                }
                render <<< blocks, threads >>> (app.devicePtr, d_gBuffer, scene.d_camera, scene.d_lightsIndex, scene.device.d_objs, scene.internalNodes, scene.lightsCount, nx, ny, sampleCount, t);
                checkCudaErrors(hipDeviceSynchronize());

                //gaussian <<< blocks, threads >>> (app.devicePtr, nx, ny);
                //hipDeviceSynchronize();

                //addPrevious << < blocks, threads >> > (app.devicePtr, d_gBuffer, nx, ny);
                //hipDeviceSynchronize();
                
                if (parser.hasCloth)
                    scene.Update();
            }

            app.Update();
            preStats = app.paused;
            preTime = t;
        }
    }
}