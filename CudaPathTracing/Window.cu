#include "hip/hip_runtime.h"

#include <iostream>
#include "Window.cuh"

Scene* Window::scene = nullptr;
float Window::alphaX = 0.5;
float Window::alphaY = 0.5;
bool Window::glass = false;
int Window::selectSampleCount = 64;
int Window::currentType = 0;
float Window::sigmaG = 8.0;
float Window::sigmaR = 5.0;
float Window::sigmaN = 0.1;
float Window::sigmaD = 0.2;

Window::Window(int w, int h, Camera* _camera, Scene* _scene) : width(w), height(h), tex(0), deltaTime(0.0),
renderType{ "RealTimeWithoutPass", "RealTime", "Normal", "Depth" }
{
    camera = _camera;
    window = nullptr;
    scene = _scene;
    sampleCount = 1;
}

Window::~Window()
{
    glDeleteTextures(1, &tex);
    glfwDestroyWindow(window);
    glfwTerminate();
}

GLuint Window::createShaderProgram()
{
    const char* vertex_shader_code = 
R"(#version 330 core
layout (location = 0) in vec3 aPos;
layout (location = 1) in vec2 aTexCoord;
out vec2 TexCoord;
void main()
{
    gl_Position = vec4(aPos, 1.0);
    TexCoord = aTexCoord;
}
)";

    const char* fragment_shader_code = 
R"(#version 330 core
in vec2 TexCoord;
uniform sampler2D tex;
out vec4 FragColor;
void main()
{
    FragColor = texture(tex, TexCoord);
}
)";

    GLint success;
    char info_log[512];

    // ������ɫ��
    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertex_shader_code, nullptr);
    glCompileShader(vertexShader);
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(vertexShader, 512, nullptr, info_log);
        std::cerr << "ERROR::VERTEX_SHADER::COMPILATION_FAILED\n" << info_log << std::endl;
    }

    // ƬԪ��ɫ��
    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragment_shader_code, nullptr);
    glCompileShader(fragmentShader);
    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(fragmentShader, 512, nullptr, info_log);
        std::cerr << "ERROR::FRAGMENT_SHADER::COMPILATION_FAILED\n" << info_log << std::endl;
    }

    // ���� shader program
    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shaderProgram, 512, nullptr, info_log);
        std::cerr << "ERROR::SHADER_PROGRAM::LINK_FAILED\n" << info_log << std::endl;
    }

    // ɾ����ʱ shader
    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    return shaderProgram;
}

void Window::setupTexturedQuad()
{
    float vertices[] = {
        // positions         // texCoords
         1.0f,  1.0f, 0.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   0.0f, 1.0f  // top left
    };
    unsigned int indices[] = {
        0, 1, 3,
        1, 2, 3
    };

    shaderProgram = createShaderProgram();
    glUseProgram(shaderProgram);
    // generate vertex array object
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    // generate vertex buffer object
    glGenBuffers(1, &VBO);
    // generate element buffer object
    glGenBuffers(1, &EBO);
    // bind the vertex array object
    glBindVertexArray(VAO);
    // copy vertices array into vertex buffer
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    // copy indices array into element buffer
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
    // configure vertex attributes
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float),
        (void*) nullptr);
    glEnableVertexAttribArray(0); // pos
    glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float),
        (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1); // texture coords

    // texture
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    // ע�� OpenGL �������� CUDA ͼ����Դ
    glGenBuffers(1, &bufferObj);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, bufferObj);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);
    hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone);

    size_t size;
    // ӳ����Դ
    hipGraphicsMapResources(1, &resource, NULL);
    hipGraphicsResourceGetMappedPointer((void**)&devicePtr, &size, resource);
    // we should immediately unmap the resource, according to the RULES
    hipGraphicsUnmapResources(1, &resource, NULL);
}

bool Window::Init()
{
    if (!glfwInit())
        return false;
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(width, height, "CUDA Path Tracing", nullptr, nullptr);
    if (!window)
    {
        glfwTerminate();
        return false;
    }

    glfwMakeContextCurrent(window);

    // ��ʼ�� GLAD
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) 
    {
        std::cerr << "GLAD ��ʼ��ʧ��\n";
        return false;
    }

    setupTexturedQuad();
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetInputMode(window, GLFW_STICKY_KEYS, GLFW_TRUE);

    // ��ʼ�� ImGui
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO();
    io.Fonts->AddFontDefault();

    ImGui::StyleColorsDark();

    // ��ʼ��ƽ̨/��Ⱦ����
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330");

    return true;
}

bool Window::Close()
{
    return glfwWindowShouldClose(window);
}

void Window::Update()
{
    glClear(GL_COLOR_BUFFER_BIT);

    // �ϴ�����
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, bufferObj);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    
    glUseProgram(shaderProgram);
    glBindVertexArray(VAO);
    glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, nullptr);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);    // ! IMPORTANT, or imgui can not display properly

    // ÿ֡��ʼ
    ImGui_ImplOpenGL3_NewFrame();
    ImGui_ImplGlfw_NewFrame();
    ImGui::NewFrame();

    // GUI ����
    ImGui::Begin("Config");
    // fps
    fpsCurve[fpsIndex] = 1.0f / deltaTime;
    fpsIndex = (fpsIndex + 1) % fpsCurveSize;
    ImGui::PlotLines("FPS", fpsCurve, fpsCurveSize, fpsIndex,
        nullptr, 0.0f, 60.0f, ImVec2(0, 100));
    // config
    ImGui::SliderFloat("alphaX", &Window::alphaX, 0.01f, 1.0f);
    ImGui::SliderFloat("alphaY", &Window::alphaY, 0.01f, 1.0f);
    ImGui::Checkbox("isGlass", &Window::glass);
    ImGui::Combo("renderType", &Window::currentType, renderType, IM_ARRAYSIZE(renderType));
    ImGui::SliderInt("spp", &Window::selectSampleCount, 1, 2048);
    // sigma
    ImGui::SliderFloat("sigmaG", &Window::sigmaG, 1.0f, 16.0f);
    ImGui::SliderFloat("sigmaR", &Window::sigmaR, 0.01f, 10.0f);
    ImGui::SliderFloat("sigmaN", &Window::sigmaN, 0.01f, 10.0f);
    ImGui::SliderFloat("sigmaD", &Window::sigmaD, 0.01f, 1.0f);
    ImGui::End();

    // ��Ⱦ
    ImGui::Render();
    ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

    glfwSwapBuffers(window);
    glfwPollEvents();
}


bool Window::PollInput()
{
    glfwPollEvents();

    if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
    {
        if (!spacePressed)
        {
            paused = !paused;
            spacePressed = true;
            sampleCount = paused ? selectSampleCount : 1;
            std::cout << (paused ? "Paused\n" : "Resumed\n");
            return true;
        }
    }
    else
    {
        spacePressed = false;
    }

    bool moved = false;
    double moveSpeed = 1.0;
    double phi = 0.0;
    double theta = 0.0;
    double x = 0.0;
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS || GLFW_RELEASE) {
        theta += moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_W);
        moved = true;
    }
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS || GLFW_RELEASE) {
        theta -= moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_S);
        moved = true;
    }
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS || GLFW_RELEASE) {
        phi -= moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_A);
        moved = true;
    }
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS || GLFW_RELEASE) {
        phi += moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_D);
        moved = true;
    }
    if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS || GLFW_RELEASE) {
        x += 4.0 * moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_Q);
        moved = true;
    }
    if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS || GLFW_RELEASE) {
        x -= 4.0 * moveSpeed * deltaTime * glfwGetKey(window, GLFW_KEY_E);
        moved = true;
    }
    if (moved)
        camera->move(phi, theta, x);
    camera->isMoving = moved;
    return moved;
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
    if (button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS)
    {
        double xpos, ypos;
        glfwGetCursorPos(window, &xpos, &ypos);
        int pixelX = static_cast<int>(xpos);
        int pixelY = static_cast<int>(ypos);
        int windowWidth, windowHeight;
        glfwGetWindowSize(window, &windowWidth, &windowHeight);
        pixelY = windowHeight - pixelY;

        getObject << <1, 1 >> > (Window::scene->device.d_objs, Window::scene->d_camera, Window::scene->internalNodes, Window::scene->d_selectPtr, pixelX, pixelY);
        hipDeviceSynchronize();

        // select object
        unsigned int selectPtr;
        hipMemcpy(&selectPtr, Window::scene->d_selectPtr, sizeof(unsigned int), hipMemcpyDeviceToHost);
        for(const auto& obj : Window::scene->objects)
        {
            if (selectPtr >= obj.beginPtr && selectPtr < obj.endPtr)
            {
                // change material
                int threadsPerBlock = 512;
                int blocks = (obj.endPtr - obj.beginPtr + threadsPerBlock - 1) / threadsPerBlock;
                changeMaterial << <blocks, threadsPerBlock >> > (Window::scene->device.d_objs, obj.beginPtr, obj.endPtr, 
                    static_cast<double>(Window::alphaX), static_cast<double>(Window::alphaY), Window::glass);
                hipDeviceSynchronize();
                std::cout << "Set " << obj.name << " Material [" << obj.beginPtr << ", " << obj.endPtr << "]" << std::endl;
                break;
            }
        }
    }
}